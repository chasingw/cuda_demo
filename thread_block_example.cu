#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_BLOVKS 16 
#define BLOCK_WIDTH 1

__global__ void hello(){

    printfg("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc, char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    hipDeviceSynchronize();
    printf("That's all!\n");

    return 0;
}
